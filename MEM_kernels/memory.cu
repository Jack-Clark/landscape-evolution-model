
#include "memory.h"


int createcontribAspace(Data* data)
{
	int fullsize;
	int dataSize;
	fullsize =  data->mapInfo.width * data->mapInfo.height;
	dataSize = fullsize * sizeof(int);
	data->contribA = (int *) malloc(dataSize);
	fprintf(data->outlog,"Host memory allocation for contribA  \n");
	return 0;
}


int clearcontribAspace(Data* data)
{
	free(data->contribA);
	//free(data->watershed_id); // need to clear this?

	return 0;
}

int createProcessMatrices(Data* data)
{
  int fullsize;
  int dataSize;
  int dataSizeInt;
  fullsize =  data->mapInfo.width * data->mapInfo.height;
  dataSize = fullsize * sizeof(double);

// these are the static grids in which data is stored from one iteration to the next ie. these are ONLY freed at the end of the simulation

  checkCudaErrors(hipHostMalloc((void **)&data->fd, sizeof(int) * data->mapInfo.height * data->mapInfo.width));
  fprintf(data->outlog, "Flow direction space on host allocated \n");

  checkCudaErrors(hipHostMalloc((void **)&data->fa,           dataSize));
  fprintf(data->outlog, "Flow accumulation space on host allocated \n");

  checkCudaErrors(hipHostMalloc((void **)&data->SlopePtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->runoffweight, dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->stonePtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->finesPtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->soilMPtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->soilBPtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->soilTPtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->nutPtr,       dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->TotBPtr,      dataSize));

  checkCudaErrors(hipHostMalloc((void **)&data->eroPtr,       dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->geliPtr,      dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->inciPtr,      dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->depoPtr,      dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->dz,           dataSize));

  checkCudaErrors(hipHostMalloc((void **)&data->weatherC,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->weatherP,     dataSize));

  fprintf(data->outlog, "All hosts matrices memory allocated \n");

  return 0;

}


int deleteProcessMatrices(Data* data)
{

	  checkCudaErrors(hipHostFree(data->dem));
	  checkCudaErrors(hipHostFree(data->fd));
	  checkCudaErrors(hipHostFree(data->fa));
	  checkCudaErrors(hipHostFree(data->SlopePtr));

	  checkCudaErrors(hipHostFree(data->runoffweight));
	  checkCudaErrors(hipHostFree(data->stonePtr));
	  checkCudaErrors(hipHostFree(data->finesPtr));
	  checkCudaErrors(hipHostFree(data->soilMPtr));
	  checkCudaErrors(hipHostFree(data->soilBPtr));
	  checkCudaErrors(hipHostFree(data->soilTPtr));
	  checkCudaErrors(hipHostFree(data->nutPtr));
	  checkCudaErrors(hipHostFree(data->TotBPtr));

	  checkCudaErrors(hipHostFree(data->eroPtr));
	  checkCudaErrors(hipHostFree(data->geliPtr));
	  checkCudaErrors(hipHostFree(data->inciPtr));
	  checkCudaErrors(hipHostFree(data->depoPtr));
	  checkCudaErrors(hipHostFree(data->dz));
	  checkCudaErrors(hipHostFree(data->weatherC));
	  checkCudaErrors(hipHostFree(data->weatherP));

	  fprintf(data->outlog, "All hosts matrices memory freed \n");

	return 0;
}


//////////////////////////////////////////////////////////////////////////////
// Setup store for catchment data ( needed for summary outputs etc)
//////////////////////////////////////////////////////////////////////////////

int createCatchmentSpace(Data* data, Catchment* Catchments) {
	//allocate space for catchment data and selective list and set values to zero
	Catchments->watershed_id = (int *) calloc(sizeof(int) , data->mapInfo.height * data->mapInfo.width);
	Catchments->mask = (int *) calloc(sizeof(int),  data->mapInfo.height * data->mapInfo.width); // all mask values set to zero

	fprintf(data->outlog, "Catchment space allocated \n");
	return 0;
}


