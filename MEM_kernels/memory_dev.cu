
#include "memory_dev.h"

void setdevicespace_FD(Data* data, Data* device)
{
	 size_t freenow, total;
	 int fullsize;
	 int ncell_x = data->mapInfo.width;
	 int ncell_y = data->mapInfo.height;
	 fullsize= ncell_x * ncell_y;

	  hipMalloc((void**) &device->fd, fullsize * sizeof(int));
	  hipMalloc((void **)&(device->dx), 9 * sizeof(int));
	  hipMalloc((void **)&(device->dy), 9 * sizeof(int));
	  hipMalloc((void **)&(device->shortest_paths), fullsize * sizeof(float));
	  hipMalloc((void**)&(device->lowHeight), fullsize * sizeof(double));
	  hipMalloc((void **) &device->watershed_id, ncell_x * ncell_y * sizeof(int));

	  hipMemGetInfo(&freenow, &total);
	  fprintf(data->outlog, "Memory on CUDA card free after FD space allocated: %d total: %d \n",freenow/1024,total/1024);
	  fprintf(data->outlog, "FD: setdevicespace:%s\n", hipGetErrorString(hipGetLastError()));

}


void cleardevicespace_FD(Data* data, Data* device)
{
	size_t freenow, total;

		hipFree(device->fd);
		hipFree(device->dx);
		hipFree(device->dy);
		hipFree(device->shortest_paths);
		hipFree(device->lowHeight);
		hipFree(device->watershed_id);
		fprintf(data->outlog, "FD: error after FD clear :%s\n", hipGetErrorString(hipGetLastError()));

		hipMemGetInfo(&freenow, &total);
		fprintf(data->outlog, "FD: Memory on CUDA card free after FD space freed: %d total: %d \n\n",freenow/1024,total/1024);

}

void setdevicespace_FA(Data* data, Data* device)
{
	int full_size;
	int ncell_x = data->mapInfo.width;
	int ncell_y = data->mapInfo.height;
	full_size= ncell_x * ncell_y;

	hipMalloc( (void**) &device->runoffweight, full_size * sizeof(double));
	hipMalloc( (void**) &device->fa, full_size * sizeof(double));
	hipMalloc((void**) &device->fd, full_size * sizeof(int));

	hipMalloc( (void**) &device->stonePtr, full_size * sizeof(double));
	hipMalloc( (void**) &device->TotBPtr, full_size * sizeof(double));
	hipMalloc( (void**) &device->soilMPtr, full_size * sizeof(double));

	// now copy the necessary data - these will not overlap becasue they are all on the same stream

	//checkCudaErrors(hipSetDevice(0));
	//checkCudaErrors( hipMemcpy( device->fa, data->fa, full_size * sizeof(double), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->fd, data->fd, full_size * sizeof(int), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->runoffweight, data->runoffweight, full_size * sizeof(double), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->stonePtr, data->stonePtr, full_size * sizeof(double), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->TotBPtr, data->TotBPtr, full_size * sizeof(double), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->soilMPtr, data->soilMPtr, full_size * sizeof(double), hipMemcpyHostToDevice)) ;

	fprintf(data->outlog, "FA: setdevicespace_FA:%s\n", hipGetErrorString(hipGetLastError()));
}

void cleardevicespace_FA(Data* data, Data* device)
{
	size_t freenow, total;

	hipFree(device->fd);
	hipFree(device->runoffweight);
	hipFree(device->fa);

	hipFree(device->contribA); // free it here as it is no longer needed)

	hipMemGetInfo(&freenow, &total);
	fprintf(data->outlog, "FA: Memory on CUDA card free after FA space freed: %d total: %d \n\n",freenow/1024,total/1024);
}


void setdevicespace_Process(Data* data, Data* device)
{
	size_t freenow, total;
	int full_size;
	int ncell_x = data->mapInfo.width;
	int ncell_y = data->mapInfo.height;
	full_size= ncell_x * ncell_y;



		hipMalloc( (void**) &device->fa,       full_size * sizeof(double));
		hipMalloc( (void**) &device->fd,       full_size * sizeof(int));
		hipMalloc( (void**) &device->dz,       full_size * sizeof(double)); // create room for product dz
		hipMalloc( (void**) &device->finesPtr, full_size * sizeof(double));
		hipMalloc( (void**) &device->soilTPtr, full_size * sizeof(double));
		hipMalloc( (void**) &device->nutPtr,   full_size * sizeof(double));
		hipMalloc( (void**) &device->soilBPtr, full_size * sizeof(double));
		hipMalloc( (void**) &device->eroPtr,   full_size * sizeof(double));
		hipMalloc( (void**) &device->geliPtr,  full_size * sizeof(double));
		hipMalloc( (void**) &device->inciPtr,  full_size * sizeof(double));
		hipMalloc( (void**) &device->depoPtr,  full_size * sizeof(double));
		hipMalloc( (void**) &device->weatherC, full_size * sizeof(double));
		hipMalloc( (void**) &device->weatherP, full_size * sizeof(double));

		fprintf(data->outlog, "MOD: setdevicespace_Process :%s\n", hipGetErrorString(hipGetLastError()));


		// stones, TotBio, soilM plus dem, slope and mask still on device
		checkCudaErrors( hipMemcpy ( device->fa,       data->fa,         full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->fd,       data->fd,         full_size * sizeof(int),    hipMemcpyHostToDevice) );

		checkCudaErrors( hipMemcpy ( device->SlopePtr,  data->SlopePtr,  full_size * sizeof(double), hipMemcpyHostToDevice) );

		checkCudaErrors( hipMemcpy ( device->finesPtr, data->finesPtr,   full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->soilTPtr, data->soilTPtr,   full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->nutPtr,   data->nutPtr,     full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->soilBPtr, data->soilBPtr,   full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->eroPtr,   data->eroPtr,     full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->geliPtr,  data->geliPtr,    full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->inciPtr,  data->inciPtr,    full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->depoPtr,  data->depoPtr,    full_size * sizeof(double), hipMemcpyHostToDevice) );

		checkCudaErrors( hipMemcpy ( device->SlopePtr,  data->SlopePtr,  full_size * sizeof(double), hipMemcpyHostToDevice) );

		fprintf(data->outlog, "MOD: Matrix memcopy operations :%s\n", hipGetErrorString(hipGetLastError()));

		hipMemGetInfo(&freenow, &total);
		fprintf(data->outlog, "MOD: Memory on CUDA card free after model matrix space allocated: %d total: %d \n",freenow/1024,total/1024);
}

void cleardevicespace_Process(Data* data, Data* device)
{
	size_t freenow, total;

	hipFree(device->fa);
	hipFree(device->fd);

	hipFree(device->dz);

	hipFree(device->finesPtr);
	hipFree(device->soilTPtr);
	hipFree(device->nutPtr);
	hipFree(device->soilBPtr);
	hipFree(device->eroPtr);
	hipFree(device->geliPtr);
	hipFree(device->inciPtr);
	hipFree(device->depoPtr);
	hipFree(device->weatherC);
	hipFree(device->weatherP);

	// free after being left at end of FA routines.
	hipFree(device->stonePtr);
	hipFree(device->TotBPtr);
	hipFree(device->soilMPtr);

	hipMemGetInfo(&freenow, &total);
	fprintf(data->outlog, "MOD: Memory on CUDA card free after model space freed: %d total: %d \n",freenow/1024,total/1024);
}

int copyMask(Data* data, Data* device)
{

	 int fullsize;
	 int ncell_x = data->mapInfo.width;
	 int ncell_y = data->mapInfo.height;
	 fullsize= ncell_x * ncell_y;

	 hipMalloc( (void**) &device->mask, fullsize * sizeof(int)); // create space for the mask

	 hipMemcpy(device->mask, data->mask, fullsize * sizeof(int), hipMemcpyHostToDevice);  // copy back flag
	 fprintf(data->outlog, "Mask data sent to device \n");

	thrust::device_ptr<int> activecells = thrust::device_pointer_cast(device->mask);
	data->activecells  = thrust::count(activecells, activecells + fullsize, 1);
	printf("No of active cells = %d \n", data->activecells);

	return 0;
}



int createDeviceSpace(Data* data, Data* device)
{
	size_t freenow, total;

	 int fullsize;
	 int ncell_x = data->mapInfo.width;
	 int ncell_y = data->mapInfo.height;
	 fullsize= ncell_x * ncell_y;

	  hipMalloc((void **)&(device->dem), fullsize* sizeof(double));
	  hipMalloc((void **)&(device->SlopePtr), fullsize * sizeof(double));

	  hipMalloc((void **)&(device->summary), fullsize * sizeof(double));

	fprintf(data->outlog,"Allocated DEM and slope matrices on device :%s\n", hipGetErrorString(hipGetLastError()));

	hipMemGetInfo(&freenow, &total);
	fprintf(data->outlog,"Memory on CUDA card free after device DEM and slope grids allocated: %d total: %d \n",freenow/1024,total/1024);

	printf("Device space created \n");
	return 0;
}


int clearDeviceSpace(Data* data, Data* device)
{
	size_t freenow, total;

		hipFree(device->dem);
		hipFree(device->SlopePtr);
		hipFree(device->summary);


	hipMemGetInfo(&freenow, &total);
	printf("Memory on CUDA card free after DEM and slope device grids space freed: %d total: %d \n",freenow/1024,total/1024);
	fprintf(data->outlog,"Memory on CUDA card free after DEM and slope device grids space freed: %d total: %d \n",freenow/1024,total/1024);


	free(data->watershed_id);


	return 0;
}



int zerogrids(Data* data)
{

	memset(data->eroPtr, 0.0, sizeof(data->eroPtr));
	memset(data->geliPtr, 0.0, sizeof(data->eroPtr));
	memset(data->inciPtr, 0.0, sizeof(data->inciPtr));
	memset(data->depoPtr, 0.0, sizeof(data->depoPtr));

	return 0;
}
