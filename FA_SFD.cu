
#include "FA_SFD.h"

int correctflow_SFD(Data* data, Data* device, int iter, int algorithmID) {
	if (hipSuccess != hipSetDevice(CUDA_DEVICE)) {
		printf("Unable to access CUDA card\n");
		return 1;
	}

	int x;

	hipEvent_t start, stop;
	float time;
    int rows = data->mapInfo.height;
	int cols = data->mapInfo.width;
	int totalCells = rows * cols;

	for (x = 0; x < rows * cols; ++x) {
		data->fa[x] = -1.0;
	}

	fprintf(data->outlog, "FA: set fagrid values to -1\n");
	hipMemcpy(device->fa, data->fa, totalCells * sizeof(double), hipMemcpyHostToDevice);
	fprintf(data->outlog, "FA: FA memcopy operation :%s\n", hipGetErrorString(hipGetLastError()));

	fprintf(data->outlog, "FA: Calling process\n");

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	switch(algorithmID) {

		case 1:
			process_SFD_NoPart_List(data, device, iter);
			break;
	
		case 2:
			process_SFD_block_level_single_chains(data, device, iter);
			break;

		case 3:
			process_SFD_global_level_single_chains(data, device, iter);
			break;

		case 4:
			process_SFD_Multiple_Retries(data, device, iter);
			break;

		default:
			fprintf(data->outlog, "\nInvalid algorithmID. Exiting...\n");
			printf("\nInvalid algorithmID. Exiting...\n");
			return 1;
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Time to complete FA_SFD_list : %.6f s\n", time / 1000.0);

	return 0;
}
