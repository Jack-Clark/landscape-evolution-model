#include "hip/hip_runtime.h"
#include "FA_SFD.h"

__global__ void kernelfunction_SFD_Initital_Compute_Deps_And_Resolve_Zero_Deps(int *mask, int *fd, double *fa, int rows, int cols, double *weights, unsigned int* pkgprogressd, int *dep) {

	// The number of neighbour cells flowing into this cell
	int depCount = 0;

	// number of neighbours that have had their FA calculated
 	int numNeighboursReady = 0;

	int irow = blockIdx.y * blockDim.y + threadIdx.y;
	int icol = blockIdx.x * blockDim.x + threadIdx.x;
	int maxSize = rows * cols;

	if (irow >= rows || icol >= cols)
		return;

	int self = irow * cols + icol;
	if (mask[self] == 0) return; // don't calculate if not in catchment(s) of interest

	int nie, nise, nis, nisw, niw, ninw, nin, nine;

	double accum = 1.0 * weights[self];

	nie  = self        + 1 ;
	nise = self + cols + 1 ;
	nis  = self + cols     ;
	nisw = self + cols - 1 ;
	niw  = self        - 1 ;
	ninw = self - cols - 1 ;
	nin  = self - cols     ;
	nine = self - cols + 1 ;

	if (icol < cols - 1 && fd[nie] & WEST) {
		depCount++;
		if (fa[nie] >= 0) {
			numNeighboursReady++;
			accum += fa[nie];
		}
	}
	if (icol < cols - 1 && fd[nise] & NORTHWEST) {
		depCount++;
		if (fa[nise] >= 0) {
			numNeighboursReady++;
			accum += fa[nise];
		} 
	}
	if (icol < cols - 1 && fd[nis] & NORTH) {
		depCount++;
		if (fa[nis] >= 0) {
			numNeighboursReady++;
			accum += fa[nis];
		}
	}
	if (icol < cols - 1 && fd[nisw] & NORTHEAST) {
		depCount++;
		if (fa[nisw] >= 0) {
			numNeighboursReady++;
			accum += fa[nisw];
		}
	}
	if (icol < cols - 1 && fd[niw] & EAST) {
		depCount++;
		if (fa[niw] >= 0) {
			numNeighboursReady++;
			accum += fa[niw];
		}
	}
	if (icol < cols - 1 && fd[ninw] & SOUTHEAST) {
		depCount++;
		if (fa[ninw] >= 0) {
			numNeighboursReady++;
			accum += fa[ninw];
		} 
	}
	if (icol < cols - 1 && fd[nin] & SOUTH) {
		depCount++;
		if (fa[nis] >= 0) {
			numNeighboursReady++;
			accum += fa[nis];
		}
	}
	if (icol < cols - 1 && fd[nine] & SOUTHWEST) {
		depCount++;
		if (fa[nine] >= 0) {
			numNeighboursReady++;
			accum += fa[nine];
		}
	}

	if((depCount - numNeighboursReady) == 0) {
		fa[self] = accum;
	} else {
		atomicInc(pkgprogressd, maxSize);
	}
	dep[self] = depCount;
}


__global__ void kernelfunction_SFD_Resolve_Zero_Dependencies(int *mask, int *fd, double *fa, int rows, int cols, double *weights, unsigned int* pkgprogressd) {

	int irow = blockIdx.y * blockDim.y + threadIdx.y;
	int icol = blockIdx.x * blockDim.x + threadIdx.x;
	int maxSize = rows * cols;

	if (irow >= rows || icol >= cols)
		return;

	int self = irow * cols + icol;
	if (mask[self] == 0) return; // don't calculate if not in catchment(s) of interest

	if(fa[self] >= 0) return;

	int nie, nise, nis, nisw, niw, ninw, nin, nine;

	double accum = 1.0 * weights[self];

	nie  = self        + 1 ;
	nise = self + cols + 1 ;
	nis  = self + cols     ;
	nisw = self + cols - 1 ;
	niw  = self        - 1 ;
	ninw = self - cols - 1 ;
	nin  = self - cols     ;
	nine = self - cols + 1 ;

	if (icol < cols - 1 && fd[nie] & WEST) {
		if (fa[nie] < 0) {
			atomicInc(pkgprogressd, maxSize);
			return;
		}
		accum += fa[nie];
	}
	if (icol < cols - 1 && fd[nise] & NORTHWEST) {
		if (fa[nise] < 0) {
			atomicInc(pkgprogressd, maxSize);
			return;
		}
		accum += fa[nise];
	}
	if (icol < cols - 1 && fd[nis] & NORTH) {
		if (fa[nis] < 0) {
			atomicInc(pkgprogressd, maxSize);
			return;
		}
		accum += fa[nis];
	}
	if (icol < cols - 1 && fd[nisw] & NORTHEAST) {
		if (fa[nisw] < 0) {
			atomicInc(pkgprogressd, maxSize);
			return;
		}
		accum += fa[nisw];
	}
	if (icol < cols - 1 && fd[niw] & EAST) {
		if (fa[niw] < 0) {
			atomicInc(pkgprogressd, maxSize);
			return;
		}
		accum += fa[niw];
	}
	if (icol < cols - 1 && fd[ninw] & SOUTHEAST) {
		if (fa[ninw] < 0) {
			atomicInc(pkgprogressd, maxSize);
			return;
		}
		accum += fa[ninw];
	}
	if (icol < cols - 1 && fd[nin] & SOUTH) {
		if (fa[nis] < 0) {
			atomicInc(pkgprogressd, maxSize);
			return;
		}
		accum += fa[nis];
	}
	if (icol < cols - 1 && fd[nine] & SOUTHWEST) {
		if (fa[nine] < 0) {
			atomicInc(pkgprogressd, maxSize);
			return;
		}
		accum += fa[nine];
	}

	fa[self] = accum;
}


__global__ void kernelfunction_SFD_Resolve_Single_Dependencies(int *mask, int *fd, double *fa, int rows, int cols, double *weights, unsigned int* pkgprogressd, int *dep, int *neighbourOffset) {

	int irow = blockIdx.y * blockDim.y + threadIdx.y;
	int icol = blockIdx.x * blockDim.x + threadIdx.x;

	if (irow >= rows || icol >= cols)
		return;

	int self = irow * cols + icol;
	
	if (mask[self] == 0) return; // don't calculate if not in catchment(s) of interest

	if(fa[self] > 0) return;

	int nextCellInFlow = self + neighbourOffset[__ffs(fd[self]) - 1];
	int currentCell = self;

	while(dep[nextCellInFlow] == 1 && fa[nextCellInFlow] < 0) {
		fa[nextCellInFlow] = 1.0 * weights[nextCellInFlow];
		fa[nextCellInFlow] += fa[currentCell];
		currentCell = nextCellInFlow;
		nextCellInFlow = currentCell + neighbourOffset[__ffs(fd[nextCellInFlow]) - 1];
		atomicDec(pkgprogressd, 0);
	}
}



int mod_process_SFD_NoPart_List(Data* data, Data* device, int iter) {
	printf("In process\n");

    int gridRows = data->mapInfo.height;
	int gridColumns = data->mapInfo.width;
	int grid1 = gridColumns / (BLOCKCOLS );
	int grid2 = gridRows / (BLOCKROWS );
	int fullsize = gridRows * gridColumns;

	unsigned int *progress_d;
	checkCudaErrors(hipMalloc((void **) &progress_d, sizeof(unsigned int)) );
	unsigned int *progress_h = (unsigned int*) malloc(sizeof(unsigned int));
	*progress_h = 0;

	checkCudaErrors(hipMemcpy(progress_d, progress_h, sizeof(unsigned int), hipMemcpyHostToDevice));

	int *dependencyMap;
	int *neighbourOffset_d;
	int neighbourOffset_h[] = {1, gridColumns+1, gridColumns, gridColumns-1, -1, -gridColumns-1, -gridColumns, -gridColumns+1};
	checkCudaErrors(hipMalloc((void **) &dependencyMap, fullsize * sizeof(int)));
	checkCudaErrors(hipMalloc((void **) &neighbourOffset_d, sizeof(neighbourOffset_h)/sizeof(int)));

	checkCudaErrors(hipMemcpy(neighbourOffset_d, neighbourOffset_h, sizeof(neighbourOffset_h)/sizeof(int), hipMemcpyHostToDevice));

	dim3 dimGrid(grid1, grid2);
	dim3 dimBlock(BLOCKCOLS, BLOCKROWS);

	// first run a kernel to solve those cells which are on the edges and produce the first list

	//__global__ void kernelfunction_SFD_NoPart_List(int *mask, int *fd, double *fa, int rows, int cols, double *weights, unsigned int* pkgprogressd, int* left)

	kernelfunction_SFD_Initital_Compute_Deps_And_Resolve_Zero_Deps<<<dimGrid, dimBlock>>>(device->mask, device->fd, device->fa, gridRows, gridColumns, device->runoffweight, progress_d, dependencyMap);

	kernelfunction_SFD_Resolve_Single_Dependencies<<<dimGrid, dimBlock>>>(device->mask, device->fd, device->fa, gridRows, gridColumns, device->runoffweight, progress_d, dependencyMap, neighbourOffset_d);
	
	// get the size of the array
	checkCudaErrors(hipMemcpy(progress_h, progress_d, sizeof(unsigned int), hipMemcpyDeviceToHost));

	unsigned int lastTot = gridRows * gridColumns;
	unsigned int* temp = (unsigned int*) malloc(sizeof(unsigned int));

	while (*progress_h > 0) { // while the array still has elements
		
		//printf("Cells left to process = %d\n", *progress_h);
		if (*progress_h > lastTot) {
			printf("The number of incorrect cells should be coming down!\n");
			scanf("%d", &lastTot);
		}
		lastTot = *progress_h;

		// reset the value of progressed before restarting
		*temp = 0;
		checkCudaErrors(hipMemcpy(progress_d, temp, sizeof(unsigned int), hipMemcpyHostToDevice) );
		//__global__ void kernelfunction_SFD_NoPart_ListProgress(int *fd, double *fa, int rows, int cols, double *weights, unsigned int* pkgprogressd, int* left, int* had, int hadsize)
		kernelfunction_SFD_Resolve_Zero_Dependencies<<<dimGrid, dimBlock>>>(device->mask, device->fd, device->fa, gridRows, gridColumns, device->runoffweight, progress_d);

		kernelfunction_SFD_Resolve_Single_Dependencies<<<dimGrid, dimBlock>>>(device->mask, device->fd, device->fa, gridRows, gridColumns, device->runoffweight, progress_d, dependencyMap, neighbourOffset_d);

		// get the new cell count
		checkCudaErrors(hipMemcpy(progress_h, progress_d, sizeof(unsigned int), hipMemcpyDeviceToHost) );
	}

	free(temp);

	// Copy flow accumulation back
	int count = 0;

	checkCudaErrors(hipMemcpy(data->fa, device->fa, fullsize * sizeof(double),   hipMemcpyDeviceToHost));
	fprintf(data->outlog, "FA: FA memcopy back operation 3:%s\n", hipGetErrorString(hipGetLastError()));

	double FA_max;
	int FAindex = 0;
	double cpuFA_max = 0.0;

	if (iter == 1) // cpu calculation otherwise we cannot locate the outletcell index
	{
		for (int i = 0; i < gridRows; i++) {
			for (int j = 0; j < gridColumns; j++) {
				if (data->fa[i * gridColumns + j] > cpuFA_max)
					{
					cpuFA_max = data->fa[i * gridColumns + j];
					FAindex = i * gridColumns + j;
					}
			}
		}
		data->FA_max = cpuFA_max;
		data->outletcellidx = FAindex; // this is the outlet cell which will be maintained throughout the simulation
	} else // do it faster using GPU in all subsequent iterations
		{
			thrust::device_ptr<double> max_FA = thrust::device_pointer_cast(device->fa);
			FA_max = thrust::reduce(max_FA, max_FA + fullsize, (double) 0, thrust::maximum<double>());
			data->FA_max = FA_max;
		}

	fprintf(data->outlog, "FA: Maximum FA is  %.6f s\n\n", data->FA_max);
	fprintf(data->outlog, "FA: Outletcell index is  %d s\n\n", data->outletcellidx);

	printf("Maximum FA is  %.6f s\n\n", data->FA_max);

	for (int i = 0; i < gridRows; i++) {
		for (int j = 0; j < gridColumns; j++) {
				if (data->fa[i * gridColumns + j] < 0) {
				count++;
				}
		}
	}
	fprintf(data->outlog, "FA: Bad value count (i.e. not in catchment(s) = %d\n", count);

	hipFree(progress_d);
	hipFree(dependencyMap);
	hipFree(neighbourOffset_d);

	free(progress_h);


	return 1;
}

void mod_correctflow_SFD_NoPart_List(Data* data, Data* device, int iter) {
	if (hipSuccess != hipSetDevice(CUDA_DEVICE)) {
		printf("Unable to access CUDA card\n");
		return ;
	}

	int x;
	hipEvent_t start, stop;
	float time;
    int gridRows = data->mapInfo.height;
	int gridColumns = data->mapInfo.width;
	int fullsize = gridRows * gridColumns;

	// Set all values to 0.0
	for (x = 0; x < gridRows * gridColumns; ++x) {
		data->fa[x] = -1.0;
	}

	fprintf(data->outlog, "FA: set fagrid values to -1\n");
	hipMemcpy(device->fa, data->fa, fullsize * sizeof(double), hipMemcpyHostToDevice);
	fprintf(data->outlog, "FA: FA memcopy operation :%s\n", hipGetErrorString(hipGetLastError()));

	fprintf(data->outlog, "FA: Calling process\n");

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	mod_process_SFD_NoPart_List(data, device, iter);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Time to complete FA_SFD_list : %.6f s\n", time / 1000.0);
}
