
#include "FA_SFD.h"

void correctflow_SFD(Data* data, Data* device, int iter) {
	if (hipSuccess != hipSetDevice(CUDA_DEVICE)) {
		printf("Unable to access CUDA card\n");
		return ;
	}

	int x;

	hipEvent_t start, stop;
	float time;
    int rows = data->mapInfo.height;
	int cols = data->mapInfo.width;
	int totalCells = rows * cols;

	for (x = 0; x < rows * cols; ++x) {
		data->fa[x] = -1.0;
	}

	fprintf(data->outlog, "FA: set fagrid values to -1\n");
	hipMemcpy(device->fa, data->fa, totalCells * sizeof(double), hipMemcpyHostToDevice);
	fprintf(data->outlog, "FA: FA memcopy operation :%s\n", hipGetErrorString(hipGetLastError()));

	fprintf(data->outlog, "FA: Calling process\n");

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Change this function to whichever version of the algorithm you want to run.
	process_SFD_Multiple_Retries(data, device, iter);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Time to complete FA_SFD_list : %.6f s\n", time / 1000.0);
}
