
#include "FA_SFD.h"

void correctflow_SFD(Data* data, Data* device, int iter) {
	if (hipSuccess != hipSetDevice(CUDA_DEVICE)) {
		printf("Unable to access CUDA card\n");
		return ;
	}

	int x;

	hipEvent_t start, stop;
	float time;
    int gridRows = data->mapInfo.height;
	int gridColumns = data->mapInfo.width;
	int fullsize = gridRows * gridColumns;

	for (x = 0; x < gridRows * gridColumns; ++x) {
		data->fa[x] = -1.0;
	}

	fprintf(data->outlog, "FA: set fagrid values to -1\n");
	hipMemcpy(device->fa, data->fa, fullsize * sizeof(double), hipMemcpyHostToDevice);
	fprintf(data->outlog, "FA: FA memcopy operation :%s\n", hipGetErrorString(hipGetLastError()));

	fprintf(data->outlog, "FA: Calling process\n");

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Change this function to whichever version of the algorithm you want to run.
	process_SFD_Multiple_Retries(data, device, iter);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Time to complete FA_SFD_list : %.6f s\n", time / 1000.0);
}
